#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <sys/time.h>

const int N=4096;   /* Number of Vectors */

/* do not change the following function */
double rtclock()
{
   struct timezone Tzp;
   struct timeval Tp;

   int stat;
   stat = gettimeofday (&Tp, &Tzp);

   if (stat != 0) printf("Error return from gettimeofday: %d",stat);

   return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

/* Your JOB to implement the cosine GPU kernel */
__global__ void cosine(YOURJOB)
{
    /* YOUR JOB  */ 
}

int main(int argc, char*argv[])
{
    double x[N], y[N], vlen[N];  /* data structure on CPU */
    double ** sim;
	double *d_x, *d_y, *d_sim, *d_vlen;     /* data structure for GPU */
    double * gpu_sim;                       /* data structure to be dynamically allocated */
                                            /* It holds the results copied back from GPU */
	int size = N * sizeof( double );        /* N is the number of vectors (points). */ 
    int size2 = N * N * sizeof (double);    /* N*N is the 2D similarity matrix */

    /* The following 4 variables is for configuration grid size and block size */
    /* If you let THREAD_DIMY (BLOCK_DIMY) be 1, then the grid size and block size
       are 1D, otherwise the grid size is 2D (assuming THREAD_DIMX BLOCK_DIMX are not 1*/
    int NUM_OF_THREADS_IN_X_DIM_OF_A_BLOCK; 
    int NUM_OF_THREADS_IN_Y_DIM_OF_A_BLOCK;
    int NUM_OF_BLOCKS_IN_X_DIM_OF_A_GRID; 
    int NUM_OF_BLOCKS_IN_Y_DIM_OF_A_GRID;
    int i;

    /* Dynamically allocate memory for sim[N][N] on the CPU */
    sim= (double**)malloc(sizeof(double*) *N);
    for (i=0; i<N; i++)
      sim[i]=(double*)malloc(sizeof(double)*N); 
   
    /* allocation for holding gpu results */ 
    /* convert the 2D to 1D:   sim[i][j] <--> gpu_sim[i*N+j] */
    gpu_sim= (double*) malloc (size2);

	/* allocate space for device copies */
	hipMalloc( (void **) &d_x, size );
	hipMalloc( (void **) &d_y, size );
	hipMalloc( (void **) &d_vlen, size );
    hipMalloc( (void **) &d_sim, size2);

    /* initialize with random numbers */
	for( int i = 0; i < N; i++ )
	{
      x[i] = (double) rand() / (double) RAND_MAX;
      y[i] = (double) rand() / (double) RAND_MAX;
      /* The following is for calculating |a| (|b|) term */
      vlen[i] = sqrt(x[i]*x[i]+y[i]*y[i]);
	}

	/* copy inputs to device */
	hipMemcpy( d_x, x, size, hipMemcpyHostToDevice );
	hipMemcpy( d_y, y, size, hipMemcpyHostToDevice );
	hipMemcpy( d_vlen, vlen, size, hipMemcpyHostToDevice );
	hipMemcpy( d_sim, sim, size2, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
    NUM_OF_THREADS_IN_X_DIM_OF_A_BLOCK = YOURJOB;    /* You decide how many threads in X direction of a block*/
    NUM_OF_THREADS_IN_Y_DIM_OF_A_BLOCK = YOURJOB;    /* You decide how many threads in Y direction of a block*/
    NUM_OF_BLOCKS_IN_X_DIM_OF_A_GRID = YOURJOB;      /* You decide how many blocks in X direction of a grid*/
    NUM_OF_BLOCKS_IN_Y_DIM_OF_A_GRID = YOURJOB;    /* You decide how many blocks in Y direction of a grid*/
    dim3 dimBlock(NUM_OF_THREADS_IN_X_DIM_OF_A_BLOCK, NUM_OF_THREADS_IN_Y_DIM_OF_A_BLOCK, 1);
    dim3 dimGrid(NUM_OF_BLOCKS_IN_X_DIM_OF_A_GRID, NUM_OF_BLOCKS_IN_Y_DIM_OF_A_GRID, 1);
   
    /* start the timer */ 
    double start_cpu = rtclock();

	/* your job is to implement the cosine GPU kernel */
	cosine<<< dimGrid, dimBlock>>>(YOURJOB);
    /* Ensure that the CPU codes after this line wait until GPU job finishes execution  */
    hipDeviceSynchronize();

    /* end the timer */
    double end_cpu = rtclock();
    printf("total time is %lf\n",(double)(end_cpu-start_cpu));  

	/* copy result back to host */
	hipMemcpy( gpu_sim, d_sim, size2, hipMemcpyDeviceToHost );
    /* do not change the following lines */
    for (int i=0; i<N; i++)
      for (int j=0; j<N; j++)
      {
        /* calculate results on the CPU */
        sim[i][j] = (x[i]*x[j]+y[i]*y[j])/(vlen[i]*vlen[j]);      
        /* if your GPU calculation is correct, you should NOT see any printf printout */
        /* if you do, you made a mistake in the cosine GPU kernel */
        if ( (sim[i][j] - gpu_sim[i*N+j]) > 1e-5 || (gpu_sim[i*N+j] - sim[i][j] > 1e-5))
        {
            printf("CPU %f and GPU %f results do not match!\n", sim[i][j], gpu_sim[i*N+j]);
            exit(-1);
        } 
    }


	/* clean up */
	hipFree( d_x );
	hipFree( d_y );
	hipFree( d_vlen );
	hipFree( d_sim );
	
	return 0;
} /* end main */
